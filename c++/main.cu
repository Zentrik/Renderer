#include "hip/hip_runtime.h"
// clang++-15 -std=c++20 c++/main.cpp -o c++/main -Wall -Wextra -Ofast -ffast-math -fdenormal-fp-math=positive-zero -march=native -flto=full -ltbb // -Wdouble-promotion -Wimplicit-int-float-conversion
// clang++-17 -std=c++20 main.cu -o main --cuda-gpu-arch=sm_61 -Wall -Wextra -Ofast -ffast-math -fcuda-flush-denormals-to-zero -flto=full -lcudart_static -ldl -lrt -pthread -Xcuda-ptxas -v // -Wdouble-promotion -Wimplicit-int-float-conversion

// cuda-gdb build
// clang++-17 -std=c++20 main.cu -o main --cuda-gpu-arch=sm_61 -Wall -Wextra -lcudart_static -g

// to get ptx
// clang++-17 -std=c++20 main.cu --cuda-gpu-arch=sm_61 -Wall -Wextra -Ofast -ffast-math -fcuda-flush-denormals-to-zero -flto=full -emit-llvm -S
// llc-17 -mcpu=sm_61 -mattr=+ptx80 -march=nvptx64 main-cuda-nvptx64-nvidia-cuda-sm_61.bc -o main.ptx

#include "settings.hpp"

#include "header.hpp"
#include "sphere.hpp"
#include "material.hpp"
#include "ppm.hpp"
#include "camera.hpp"

const int kSphereCount = 22 * 22 + 1 + 3;
__device__ __constant__ float4 centre_radius[kSphereCount];
__device__ __constant__ Material::MaterialType material_type[kSphereCount];
__device__ __constant__ float4 material_data[kSphereCount];
__device__ __constant__ i32 spheres_count;

#include "hittable_list.hpp"

struct BufferData {
    Ray ray;
    colour attenuation;
    u32 pixel_index;
    u32 depth;
};

class BufferDataVec {
    __device__ BufferDataVec(Ray* ray, float4* attenuation_and_pixel_index, u32* depth) : ray(ray), attenuation_and_pixel_index(attenuation_and_pixel_index), depth(depth) {}
public:
    Ray* ray;
    float4* attenuation_and_pixel_index;
    u32* depth;

    BufferDataVec(i32 state_size) {
        checkCudaErrors(hipMalloc(&ray, state_size * sizeof(Ray)));
        checkCudaErrors(hipMalloc(&attenuation_and_pixel_index, state_size * sizeof(float4)));
        checkCudaErrors(hipMalloc(&depth, state_size * sizeof(u32)));
    }
    void free () {
        // clean up
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipFree(ray));
        checkCudaErrors(hipFree(attenuation_and_pixel_index));
        checkCudaErrors(hipFree(depth));
    }

    __device__ BufferData operator[](i32 i) const {
        float4 a_p = attenuation_and_pixel_index[i];
        colour attenuation(a_p.x, a_p.y, a_p.z);
        u32 pixel_index = __float_as_uint(a_p.w);

        return BufferData(ray[i], attenuation, pixel_index, depth[i]);
    }

    __device__ BufferDataVec operator[](i32 i) {
        return BufferDataVec(&ray[i], &attenuation_and_pixel_index[i], &depth[i]);
    }

    __device__ BufferDataVec& operator=(BufferData x) {
        *ray = x.ray;
        *attenuation_and_pixel_index = make_float4(x.attenuation, __uint_as_float(x.pixel_index));
        *depth = x.depth;
        return *this;
    }
};

__device__ colour world_colour(Ray ray) {
    f32 t = (ray.direction.z + 1) / 2.f;
    return (1.f - t) * colour(1, 1, 1) + t * colour(0.5, 0.7, 1);
}

HittableList random_scene() {
    HittableList spheres;
    RNG rng{0};

    spheres.emplace_back(vec3(0, 0, -1000), 1000, Material::Lambertian(colour(0.5f, 0.5f, 0.5f)));

    for (f32 a = -11; a < 11; a++) {
        for (f32 b = -11; b < 11; b++) {
            f32 choose_mat = rng.f32();
            vec3 center(a + 0.9f * rng.f32(), -(b + 0.9f * rng.f32()), 0.2f);

            if (length(center - vec3(4, 0, 0.2f)) > 0.9f) {
                if (choose_mat < 0.8f) {
                    // diffuse
                    colour albedo = rng.colour()* rng.colour();
                    spheres.emplace_back(center, 0.2f, Material::Lambertian(albedo));
                }
                else if (choose_mat < 0.95f) {
                    // metal
                    colour albedo = rng.colour() / 2 + colour(.5f, .5f, .5f);
                    f32 fuzz = rng.f32() / 2;
                    spheres.emplace_back(center, 0.2f, Material::Metal(albedo, fuzz));
                }
                else {
                    // glass
                    spheres.emplace_back(center, 0.2f, Material::Dielectric());
                }
            }
        }
    }

    spheres.emplace_back(vec3(0, 0, 1), 1.0, Material::Dielectric());

    spheres.emplace_back(vec3(-4, 0, 1), 1.0, Material::Lambertian({0.4f, 0.2f, 0.1f}));

    spheres.emplace_back(vec3(4, 0, 1), 1.0, Material::Metal({0.7f, 0.6f, 0.5f}, 0));

    return spheres;
}

__global__ void generate_rays(BufferDataVec current_state, Camera camera, u32 column_size, i32 current_state_size, u32 offset, u32 samples_per_pixel, i32 index_offset) {
    i32 index = blockIdx.x * blockDim.x + threadIdx.x;

    for (i32 i = index; i < current_state_size; i += gridDim.x * blockDim.x) {
        u32 img_linear_index = u32((i + offset) / samples_per_pixel);

        u32 y = img_linear_index / column_size;
        u32 x = img_linear_index % column_size;

        RNG rng((1u+img_linear_index) * ((1u+i) + offset));
        Ray ray = camera.get_ray((f32)x, (f32)y, rng);

        current_state.ray[i + index_offset] = ray;
        current_state.attenuation_and_pixel_index[i + index_offset] = make_float4(1, 1, 1, __uint_as_float(img_linear_index));
        current_state.depth[i + index_offset] = 1u;
    }
}

__device__ void scatter(colour* img, BufferDataVec next_state, BufferData current_state, i32* next_state_index, RNG& rng, HitRecord hit_record, u32 max_depth) {
    u32 pixel_index = current_state.pixel_index;
    Ray r = current_state.ray;

    if (!hit(hit_record)) { // nothing hit
        colour new_attenuation = current_state.attenuation * world_colour(r);
        // printf("%f [%f, %f, %f] \n", r.direction.z, new_attenuation.x, new_attenuation.y, new_attenuation.z);
        // printf("%u \n", pixel_index);
        // new_attenuation.x = 10.f;
        atomicAdd(&(img[pixel_index].x), new_attenuation.x);
        atomicAdd(&(img[pixel_index].y), new_attenuation.y);
        atomicAdd(&(img[pixel_index].z), new_attenuation.z);
    } else {
        vec3 position = r.at(hit_record.t);

        float4 c_r = centre_radius[hit_record.sphere_index];
        vec3 centre(c_r.x, c_r.y, c_r.z);
        float radius = c_r.w;

        vec3 normal = sphere_normal(centre, radius, position);

        Material material(material_data[hit_record.sphere_index], material_type[hit_record.sphere_index]);

        auto tmp = material.scatter(r, normal, rng); // structured binding doesn't work well with gdb
        vec3 direction = std::get<0>(tmp);
        colour attenuation = std::get<1>(tmp);
        bool scatter_again = std::get<2>(tmp);

        if (scatter_again) {
            colour new_attenuation = current_state.attenuation * attenuation;

            if (current_state.depth == max_depth) {
                atomicAdd(&(img[pixel_index].x), new_attenuation.x);
                atomicAdd(&(img[pixel_index].y), new_attenuation.y);
                atomicAdd(&(img[pixel_index].z), new_attenuation.z);
            } else {
                i32 old_index = atomicAdd(next_state_index, 1);

                next_state[old_index] = BufferData(Ray(position, direction), new_attenuation, pixel_index, current_state.depth + 1u);
            }
        }
    }
}

__global__ void intersect_and_scatter(colour* img, BufferDataVec next_state, const BufferDataVec current_state, u32 max_depth, i32* next_state_index, i32 current_state_size, f32 tmin, f32 tmax, u32 number_of_rays_generated) {
    i32 index = blockIdx.x * blockDim.x + threadIdx.x;

    for (i32 i = index; i < current_state_size; i += gridDim.x * blockDim.x) {
        BufferData state = current_state[i];

        HitRecord hit_record = hit(state.ray, tmin, tmax);
        RNG rng((1u + state.pixel_index) * ((1u + i) + number_of_rays_generated) + state.depth);
        scatter(img, next_state, state, next_state_index, rng, hit_record, max_depth);
    }
}

__global__ void generate_intersect_and_scatter(colour* img, BufferDataVec next_state, u32 max_depth, i32* next_state_index, i32 rays_size, f32 tmin, f32 tmax, Camera camera, u32 offset, u32 samples_per_pixel, u32 column_size) {
    i32 index = blockIdx.x * blockDim.x + threadIdx.x;

    for (i32 i = index; i < rays_size; i += gridDim.x * blockDim.x) {
        u32 img_linear_index = (i + offset) / samples_per_pixel;

        u32 y = img_linear_index / column_size;
        u32 x = img_linear_index % column_size;

        RNG rng((1u + img_linear_index) * ((i + 1u) + offset));
        Ray ray = camera.get_ray((f32)x, (f32)y, rng);
        // printf("[%u, %u] \n", x, y);
        // printf("%u \n", img_linear_index);
        // printf("[%f, %f, %f] \n", ray.origin.x, ray.origin.y, ray.origin.z);
        // printf("[%f, %f, %f] \n", ray.direction.x, ray.direction.y, ray.direction.z);
        BufferData current_state(ray, colour(1, 1, 1), img_linear_index, 1);

        HitRecord hit_record = hit(ray, tmin, tmax);
        // printf("[%f, %u] \n", hit_record.t, hit_record.sphere_index);
        scatter(img, next_state, current_state, next_state_index, rng, hit_record, max_depth);
    }
}

i32 main() {
    // IMAGE

#ifdef PROFVIEW
    const f32 aspect_ratio = 16.f / 9;
    const i32 image_width = 1920 / 2;
    const i32 image_height = static_cast<i32>(image_width / aspect_ratio);
    const i32 samples_per_pixel = 10;
    const i32 max_depth = 16;
#else
#ifdef CLAFORTE
    const f32 aspect_ratio = 16.f / 9;
    const i32 image_width = 1920;
    const i32 image_height = static_cast<i32>(image_width / aspect_ratio);
    const i32 samples_per_pixel = 1000;
    const i32 max_depth = 16;
#else
#if 1
    f32 aspect_ratio = 16.f / 9;
    const i32 image_width = 10;
    const i32 image_height = static_cast<i32>(roundf32(image_width / aspect_ratio));
    aspect_ratio = (f32)image_width / (f32)image_height;
    const i32 samples_per_pixel = 1;
    const i32 max_depth = 16;
#else
    const f32 aspect_ratio = 16.f / 9;
    const i32 image_width = 800;
    const i32 image_height = static_cast<i32>(image_width / aspect_ratio);
    const i32 samples_per_pixel = 50;
    const i32 max_depth = 10;
#endif
#endif
#endif

    // WORLD
    HittableList world = random_scene();

    if (world.centre_radius.size() > kSphereCount) {
        std::cerr << "Too many spheres";
        return 1;
    }

    // copy the scene to constant memory
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(centre_radius), world.centre_radius.data(), world.centre_radius.size() * sizeof(float4)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(material_data), world.material_data.data(), world.material_data.size() * sizeof(float4)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(material_type), world.material_type.data(), world.material_type.size() * sizeof(u32)));
    i32 size = world.centre_radius.size();
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(spheres_count), &size, sizeof(i32)));

    // Camera

    vec3 lookfrom(13, -3, 2);
    vec3 lookat(0, 0, 0);
    vec3 vup(0, 0, 1);
    f32 dist_to_focus = 10;
    f32 aperture = 0.05;

    Camera camera((f32)image_width, (f32)image_height, lookfrom, lookat, vup, 20, aperture, dist_to_focus);

    // Render

    f32 tmin = 1e-4f;
    f32 tmax = infinity;

    colour* d_img;
    checkCudaErrors(hipMalloc(&d_img, image_height*image_width * sizeof(colour)));

    float elapsedTime;
    hipEvent_t start;
    hipEvent_t end;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&end));
    checkCudaErrors(hipEventRecord(start));

    // Render Loop Started

    i32 number_of_rays = samples_per_pixel * image_height * image_width;

    i32 max_state_size = pow(10, 7);
    i32 state_size = min(number_of_rays, max_state_size);

    BufferDataVec current_state(state_size);
    BufferDataVec next_state(state_size);

    u32 number_of_rays_generated = 0;

    i32 current_state_size = min(number_of_rays - (i32)number_of_rays_generated, state_size);
    i32* next_state_index;
    checkCudaErrors(hipMalloc(&next_state_index, sizeof(i32)));
    checkCudaErrors(hipMemset(next_state_index, 0, sizeof(i32)));

    int generate_intersect_and_scatter_blocks;
    int generate_intersect_and_scatter_threads;

    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
                &generate_intersect_and_scatter_blocks, &generate_intersect_and_scatter_threads, generate_intersect_and_scatter, 0, current_state_size));

    generate_intersect_and_scatter_threads = min(generate_intersect_and_scatter_threads, current_state_size);
    generate_intersect_and_scatter_blocks = (current_state_size + generate_intersect_and_scatter_threads - 1) / generate_intersect_and_scatter_threads;

    generate_intersect_and_scatter<<<generate_intersect_and_scatter_blocks, generate_intersect_and_scatter_threads>>>(d_img, current_state, max_depth, next_state_index, current_state_size, tmin, tmax, camera, number_of_rays_generated, samples_per_pixel, image_height);
    checkCudaErrors(hipGetLastError());

    number_of_rays_generated += current_state_size;

    checkCudaErrors(hipMemcpy(&current_state_size, next_state_index, sizeof(i32), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemset(next_state_index, 0, sizeof(i32)));

    int intersect_and_scatter_minGridSize;
    int intersect_and_scatter_blockSize;

    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
                &intersect_and_scatter_minGridSize, &intersect_and_scatter_blockSize, intersect_and_scatter, 0, current_state_size));

    int intersect_and_scatter_blocks;
    int intersect_and_scatter_threads;

    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
                &intersect_and_scatter_blocks, &intersect_and_scatter_threads, intersect_and_scatter, 0, 0));

    int generate_rays_blocks;
    int generate_rays_threads;

    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
                &generate_rays_blocks, &generate_rays_threads, generate_rays, 0, 0));

    while (current_state_size > 0) {
        i32 free_slots = min(number_of_rays - (i32)number_of_rays_generated, state_size - current_state_size);

        if (free_slots > 0) {
            generate_rays_threads = min(generate_rays_threads, current_state_size);
            generate_rays_blocks = (current_state_size + generate_rays_threads - 1) / generate_rays_threads;

            generate_rays<<<generate_rays_blocks, generate_rays_threads>>>(current_state, camera, image_height, free_slots, number_of_rays_generated, samples_per_pixel, current_state_size);
            checkCudaErrors(hipGetLastError());

            number_of_rays_generated += free_slots;
            current_state_size += free_slots;
        }

        intersect_and_scatter_threads = min(intersect_and_scatter_threads, current_state_size);
        intersect_and_scatter_blocks = (current_state_size + intersect_and_scatter_threads - 1) / intersect_and_scatter_threads;

        intersect_and_scatter<<<intersect_and_scatter_blocks, intersect_and_scatter_threads>>>(d_img, next_state, current_state, max_depth, next_state_index, current_state_size, tmin, tmax, number_of_rays_generated);
        checkCudaErrors(hipGetLastError());

        // std::swap(current_state, d_next_state);
        BufferDataVec tmp = current_state;
        current_state = next_state;
        next_state = tmp;

        checkCudaErrors(hipMemcpy(&current_state_size, next_state_index, sizeof(i32), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemset(next_state_index, 0, sizeof(i32)));
    }

    // Render Loop Ended
    checkCudaErrors(hipEventRecord(end));
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, end));

    std::cout << "\nDone in " << elapsedTime << " milliseconds\n";

    colour* h_img = new colour[image_height * image_width];
    checkCudaErrors(hipMemcpy(h_img, d_img, image_height * image_width * sizeof(colour), hipMemcpyDeviceToHost));

    std::ofstream myfile;
    myfile.open("image.ppm");

    myfile << "P3\n" << image_width << " " << image_height << "\n255\n";
    for (int j = 0; j < image_height; j++) {
        for (int i = 0; i < image_width; i++) {
            write_colour(myfile, h_img[i*image_height + j], samples_per_pixel);
        }
    }
    delete[] h_img;

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_img));
    checkCudaErrors(hipFree(next_state_index));

    current_state.free();
    next_state.free();

    checkCudaErrors(hipGetLastError());

    hipDeviceReset();
}