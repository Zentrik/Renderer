#include "hip/hip_runtime.h"
// clang++-15 -std=c++20 c++/main.cpp -o c++/main -Wall -Wextra -Ofast -ffast-math -fdenormal-fp-math=positive-zero -march=native -flto=full -ltbb // -Wdouble-promotion -Wimplicit-int-float-conversion
// clang++-17 -std=c++20 main.cu -o main --cuda-gpu-arch=sm_61 -Wall -Wextra -Ofast -ffast-math -fcuda-flush-denormals-to-zero -flto=full -lcudart_static -ldl -lrt -pthread -Xcuda-ptxas -v // -Wdouble-promotion -Wimplicit-int-float-conversion

// cuda-gdb build
// clang++-17 -std=c++20 main.cu -o main --cuda-gpu-arch=sm_61 -Wall -Wextra -lcudart_static -g

// to get ptx
// clang++-17 -std=c++20 main.cu --cuda-gpu-arch=sm_61 -Wall -Wextra -Ofast -ffast-math -fcuda-flush-denormals-to-zero -flto=full -emit-llvm -S
// llc-17 -mcpu=sm_61 -mattr=+ptx80 -march=nvptx64 main-cuda-nvptx64-nvidia-cuda-sm_61.bc -o main.ptx

#include "settings.hpp"

#include "header.hpp"
#include "sphere.hpp"
#include "material.hpp"
#include "ppm.hpp"
#include "camera.hpp"

const int kSphereCount = 22 * 22 + 1 + 3;
__device__ __constant__ float4 centre_radius[kSphereCount];
__device__ __constant__ Material::MaterialType material_type[kSphereCount];
__device__ __constant__ float4 material_data[kSphereCount];
__device__ __constant__ i32 spheres_count;

#include "hittable_list.hpp"

struct BufferData {
    Ray ray;
    colour attenuation;
    u32 pixel_index;
    u32 depth;
};

class BufferDataVec {
    __device__ BufferDataVec(Ray* ray, float4* attenuation_and_pixel_index, u32* depth) : ray(ray), attenuation_and_pixel_index(attenuation_and_pixel_index), depth(depth) {}
public:
    Ray* ray;
    float4* attenuation_and_pixel_index;
    u32* depth;

    BufferDataVec(i32 state_size) {
        checkCudaErrors(hipMalloc(&ray, state_size * sizeof(Ray)));
        checkCudaErrors(hipMalloc(&attenuation_and_pixel_index, state_size * sizeof(float4)));
        checkCudaErrors(hipMalloc(&depth, state_size * sizeof(u32)));
    }
    void free () {
        // clean up
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipFree(ray));
        checkCudaErrors(hipFree(attenuation_and_pixel_index));
        checkCudaErrors(hipFree(depth));
    }

    __device__ BufferData operator[](i32 i) const {
        float4 a_p = attenuation_and_pixel_index[i];
        colour attenuation(a_p.x, a_p.y, a_p.z);
        u32 pixel_index = __float_as_uint(a_p.w);

        return BufferData(ray[i], attenuation, pixel_index, depth[i]);
    }

    __device__ BufferDataVec operator[](i32 i) {
        return BufferDataVec(&ray[i], &attenuation_and_pixel_index[i], &depth[i]);
    }

    __device__ BufferDataVec& operator=(BufferData x) {
        *ray = x.ray;
        *attenuation_and_pixel_index = make_float4(x.attenuation, __uint_as_float(x.pixel_index));
        *depth = x.depth;
        return *this;
    }
};

void swap(BufferDataVec& x, BufferDataVec& y) {
    std::swap(x.ray, y.ray);
    std::swap(x.attenuation_and_pixel_index, y.attenuation_and_pixel_index);
    std::swap(x.depth, y.depth);
}

template<typename T>
class AtomicGPUFixedSizeVectorWrapper { // capacity set by ctor
public:
    T data;
    i32* length;

    // allocate uninitialised data, reserve T
    __host__ AtomicGPUFixedSizeVectorWrapper(i32 size) : data(size) {
        checkCudaErrors(hipMalloc(&length, sizeof(i32)));
        checkCudaErrors(hipMemset(length, 0, sizeof(i32)));
    }

    __device__ void push_back(auto x) {
        i32 index = atomicAdd(length, 1);
        data[index] = x;
    }

    __device__ auto operator[](i32 i) const {
        return data[i];
    }

    __device__ auto operator[](i32 i) {
        return data[i];
    }

    void clear() {
        checkCudaErrors(hipMemset(length, 0, sizeof(i32)));
    }

    void free () {
        // clean up
        data.free();
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipFree(length));
    }
};

template<typename T>
class GPUFixedSizeVectorWrapper { // capacity set by ctor
public:
    T data;
    i32 length;

    // allocate uninitialised data, reserve T
    __host__ GPUFixedSizeVectorWrapper(i32 size) : data(size), length(0) {}

    __device__ auto operator[](i32 i) const {
        return data[i];
    }

    __device__ auto operator[](i32 i) {
        return data[i];
    }

    __host__ __device__ i32 size() const {
        return length;
    }

    bool empty() const {
        return length == 0;
    }

    void free () {
        // clean up
        data.free();
    }
};

template<typename T>
void swap(GPUFixedSizeVectorWrapper<T>& x, AtomicGPUFixedSizeVectorWrapper<T>& y) {
    swap(x.data, y.data);
    int x_length = x.length;
    checkCudaErrors(hipMemcpy(&x.length, y.length, sizeof(i32), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemset(y.length, x_length, sizeof(i32)));
}

__device__ colour world_colour(Ray ray) {
    f32 t = (ray.direction.z + 1) / 2.f;
    return (1.f - t) * colour(1, 1, 1) + t * colour(0.5, 0.7, 1);
}

HittableList random_scene() {
    HittableList spheres;
    RNG rng{0};

    spheres.emplace_back(vec3(0, 0, -1000), 1000, Material::Lambertian(colour(0.5f, 0.5f, 0.5f)));

    for (f32 a = -11; a < 11; a++) {
        for (f32 b = -11; b < 11; b++) {
            f32 choose_mat = rng.f32();
            vec3 center(a + 0.9f * rng.f32(), -(b + 0.9f * rng.f32()), 0.2f);

            if (length(center - vec3(4, 0, 0.2f)) > 0.9f) {
                if (choose_mat < 0.8f) {
                    // diffuse
                    colour albedo = rng.colour()* rng.colour();
                    spheres.emplace_back(center, 0.2f, Material::Lambertian(albedo));
                }
                else if (choose_mat < 0.95f) {
                    // metal
                    colour albedo = rng.colour() / 2 + colour(.5f, .5f, .5f);
                    f32 fuzz = rng.f32() / 2;
                    spheres.emplace_back(center, 0.2f, Material::Metal(albedo, fuzz));
                }
                else {
                    // glass
                    spheres.emplace_back(center, 0.2f, Material::Dielectric());
                }
            }
        }
    }

    spheres.emplace_back(vec3(0, 0, 1), 1.0, Material::Dielectric());

    spheres.emplace_back(vec3(-4, 0, 1), 1.0, Material::Lambertian({0.4f, 0.2f, 0.1f}));

    spheres.emplace_back(vec3(4, 0, 1), 1.0, Material::Metal({0.7f, 0.6f, 0.5f}, 0));

    return spheres;
}

__global__ void generate_rays(GPUFixedSizeVectorWrapper<BufferDataVec> current_state, Camera camera, u32 column_size, i32 rays_to_generate, u32 offset, u32 samples_per_pixel) {
    i32 index = blockIdx.x * blockDim.x + threadIdx.x;

    for (i32 i = index; i < rays_to_generate; i += gridDim.x * blockDim.x) {
        u32 img_linear_index = u32((i + offset) / samples_per_pixel);

        u32 y = img_linear_index / column_size;
        u32 x = img_linear_index % column_size;

        RNG rng((1u+img_linear_index) * ((1u+i) + offset));
        Ray ray = camera.get_ray((f32)x, (f32)y, rng);

        current_state[i + current_state.size()] = BufferData(ray, colour(1, 1, 1), img_linear_index, 1u);
    }
}

__device__ void scatter(colour* img, AtomicGPUFixedSizeVectorWrapper<BufferDataVec> next_state, BufferData current_state, RNG& rng, HitRecord hit_record, u32 max_depth) {
    u32 pixel_index = current_state.pixel_index;
    Ray r = current_state.ray;

    if (!hit(hit_record)) { // nothing hit
        colour new_attenuation = current_state.attenuation * world_colour(r);
        // printf("%f [%f, %f, %f] \n", r.direction.z, new_attenuation.x, new_attenuation.y, new_attenuation.z);
        // printf("%u \n", pixel_index);
        // new_attenuation.x = 10.f;
        atomicAdd(&(img[pixel_index].x), new_attenuation.x);
        atomicAdd(&(img[pixel_index].y), new_attenuation.y);
        atomicAdd(&(img[pixel_index].z), new_attenuation.z);
    } else {
        vec3 position = r.at(hit_record.t);

        float4 c_r = centre_radius[hit_record.sphere_index];
        vec3 centre(c_r.x, c_r.y, c_r.z);
        float radius = c_r.w;

        vec3 normal = sphere_normal(centre, radius, position);

        Material material(material_data[hit_record.sphere_index], material_type[hit_record.sphere_index]);

        auto tmp = material.scatter(r, normal, rng); // structured binding doesn't work well with gdb
        vec3 direction = std::get<0>(tmp);
        colour attenuation = std::get<1>(tmp);
        bool scatter_again = std::get<2>(tmp);

        if (scatter_again) {
            colour new_attenuation = current_state.attenuation * attenuation;

            if (current_state.depth == max_depth) {
                atomicAdd(&(img[pixel_index].x), new_attenuation.x);
                atomicAdd(&(img[pixel_index].y), new_attenuation.y);
                atomicAdd(&(img[pixel_index].z), new_attenuation.z);
            } else {
                next_state.push_back(BufferData(Ray(position, direction), new_attenuation, pixel_index, current_state.depth + 1u));
            }
        }
    }
}

__global__ void intersect_and_scatter(colour* img, AtomicGPUFixedSizeVectorWrapper<BufferDataVec> next_state, const GPUFixedSizeVectorWrapper<BufferDataVec> current_state, u32 max_depth, f32 tmin, f32 tmax, u32 number_of_rays_generated) {
    i32 index = blockIdx.x * blockDim.x + threadIdx.x;

    for (i32 i = index; i < current_state.size(); i += gridDim.x * blockDim.x) {
        BufferData state = current_state[i];

        HitRecord hit_record = hit(state.ray, tmin, tmax);
        RNG rng((1u + state.pixel_index) * ((1u + i) + number_of_rays_generated) + state.depth);
        scatter(img, next_state, state, rng, hit_record, max_depth);
    }
}

__global__ void generate_intersect_and_scatter(colour* img, AtomicGPUFixedSizeVectorWrapper<BufferDataVec> next_state, i32 next_state_capactity, u32 max_depth, f32 tmin, f32 tmax, Camera camera, u32 offset, u32 samples_per_pixel, u32 column_size) {
    i32 index = blockIdx.x * blockDim.x + threadIdx.x;

    for (i32 i = index; i < next_state_capactity; i += gridDim.x * blockDim.x) {
        u32 img_linear_index = (i + offset) / samples_per_pixel;

        u32 y = img_linear_index / column_size;
        u32 x = img_linear_index % column_size;

        RNG rng((1u + img_linear_index) * ((i + 1u) + offset));
        Ray ray = camera.get_ray((f32)x, (f32)y, rng);
        // printf("[%u, %u] \n", x, y);
        // printf("%u \n", img_linear_index);
        // printf("[%f, %f, %f] \n", ray.origin.x, ray.origin.y, ray.origin.z);
        // printf("[%f, %f, %f] \n", ray.direction.x, ray.direction.y, ray.direction.z);
        BufferData current_state(ray, colour(1, 1, 1), img_linear_index, 1);

        HitRecord hit_record = hit(ray, tmin, tmax);
        // printf("[%f, %u] \n", hit_record.t, hit_record.sphere_index);
        scatter(img, next_state, current_state, rng, hit_record, max_depth);
    }
}

i32 main() {
    // IMAGE

#ifdef PROFVIEW
    const f32 aspect_ratio = 16.f / 9;
    const i32 image_width = 1920 / 2;
    const i32 image_height = static_cast<i32>(image_width / aspect_ratio);
    const i32 samples_per_pixel = 10;
    const i32 max_depth = 16;
#else
#ifdef CLAFORTE
    const f32 aspect_ratio = 16.f / 9;
    const i32 image_width = 1920;
    const i32 image_height = static_cast<i32>(image_width / aspect_ratio);
    const i32 samples_per_pixel = 1000;
    const i32 max_depth = 16;
#else
#if 1
    f32 aspect_ratio = 16.f / 9;
    const i32 image_width = 10;
    const i32 image_height = static_cast<i32>(roundf32(image_width / aspect_ratio));
    aspect_ratio = (f32)image_width / (f32)image_height;
    const i32 samples_per_pixel = 1;
    const i32 max_depth = 16;
#else
    const f32 aspect_ratio = 16.f / 9;
    const i32 image_width = 800;
    const i32 image_height = static_cast<i32>(image_width / aspect_ratio);
    const i32 samples_per_pixel = 50;
    const i32 max_depth = 10;
#endif
#endif
#endif

    // WORLD
    HittableList world = random_scene();

    if (world.centre_radius.size() > kSphereCount) {
        std::cerr << "Too many spheres";
        return 1;
    }

    // copy the scene to constant memory
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(centre_radius), world.centre_radius.data(), world.centre_radius.size() * sizeof(float4)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(material_data), world.material_data.data(), world.material_data.size() * sizeof(float4)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(material_type), world.material_type.data(), world.material_type.size() * sizeof(u32)));
    i32 size = world.centre_radius.size();
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(spheres_count), &size, sizeof(i32)));

    // Camera

    vec3 lookfrom(13, -3, 2);
    vec3 lookat(0, 0, 0);
    vec3 vup(0, 0, 1);
    f32 dist_to_focus = 10;
    f32 aperture = 0.05;

    Camera camera((f32)image_width, (f32)image_height, lookfrom, lookat, vup, 20, aperture, dist_to_focus);

    // Render

    f32 tmin = 1e-4f;
    f32 tmax = infinity;

    colour* d_img;
    checkCudaErrors(hipMalloc(&d_img, image_height*image_width * sizeof(colour)));

    float elapsedTime;
    hipEvent_t start;
    hipEvent_t end;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&end));
    checkCudaErrors(hipEventRecord(start));

    // Render Loop Started

    const i32 no_of_pixels = image_height * image_width;
    const i32 number_of_rays = samples_per_pixel * no_of_pixels;

    const i32 max_state_size = pow(10, 7);
    const i32 state_size = min(number_of_rays, max_state_size);
    i32 number_of_rays_generated = 0;

    GPUFixedSizeVectorWrapper<BufferDataVec> current_state(state_size);
    AtomicGPUFixedSizeVectorWrapper<BufferDataVec> next_state(state_size);

    int generate_intersect_and_scatter_blocks;
    int generate_intersect_and_scatter_threads;

    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
                &generate_intersect_and_scatter_blocks, &generate_intersect_and_scatter_threads, generate_intersect_and_scatter, 0, 0));

    generate_intersect_and_scatter_threads = min(generate_intersect_and_scatter_threads, state_size);
    generate_intersect_and_scatter_blocks = (state_size + generate_intersect_and_scatter_threads - 1) / generate_intersect_and_scatter_threads;

    generate_intersect_and_scatter<<<generate_intersect_and_scatter_blocks, generate_intersect_and_scatter_threads>>>(d_img, next_state, state_size, max_depth, tmin, tmax, camera, number_of_rays_generated, samples_per_pixel, image_height);
    checkCudaErrors(hipGetLastError());

    number_of_rays_generated = state_size;

    swap(current_state, next_state);
    next_state.clear();

    int intersect_and_scatter_blocks;
    int intersect_and_scatter_threads;

    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
                &intersect_and_scatter_blocks, &intersect_and_scatter_threads, intersect_and_scatter, 0, 0));

    int generate_rays_blocks;
    int generate_rays_threads;

    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
                &generate_rays_blocks, &generate_rays_threads, generate_rays, 0, 0));

    while (!current_state.empty()) {
        i32 rays_to_generate = min(number_of_rays - number_of_rays_generated, state_size - current_state.size());

        if (rays_to_generate > 0) {
            generate_rays_threads = min(generate_rays_threads, rays_to_generate);
            generate_rays_blocks = (rays_to_generate + generate_rays_threads - 1) / generate_rays_threads;

            generate_rays<<<generate_rays_blocks, generate_rays_threads>>>(current_state, camera, image_height, rays_to_generate, number_of_rays_generated, samples_per_pixel);
            checkCudaErrors(hipGetLastError());

            number_of_rays_generated += rays_to_generate;
            current_state.length += rays_to_generate;
        }

        intersect_and_scatter_threads = min(intersect_and_scatter_threads, current_state.size());
        intersect_and_scatter_blocks = (current_state.size() + intersect_and_scatter_threads - 1) / intersect_and_scatter_threads;

        intersect_and_scatter<<<intersect_and_scatter_blocks, intersect_and_scatter_threads>>>(d_img, next_state, current_state, max_depth, tmin, tmax, number_of_rays_generated);
        checkCudaErrors(hipGetLastError());

        swap(current_state, next_state);
        next_state.clear();
    }

    // Render Loop Ended
    checkCudaErrors(hipEventRecord(end));
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, end));

    std::cout << "\nDone in " << elapsedTime << " milliseconds\n";

    colour* h_img = new colour[no_of_pixels];
    checkCudaErrors(hipMemcpy(h_img, d_img, no_of_pixels * sizeof(colour), hipMemcpyDeviceToHost));

    std::ofstream myfile;
    myfile.open("image.ppm");

    myfile << "P3\n" << image_width << " " << image_height << "\n255\n";
    for (int j = 0; j < image_height; j++) {
        for (int i = 0; i < image_width; i++) {
            write_colour(myfile, h_img[i*image_height + j], samples_per_pixel);
        }
    }
    delete[] h_img;

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_img));

    current_state.free();
    next_state.free();

    checkCudaErrors(hipGetLastError());

    hipDeviceReset();
}